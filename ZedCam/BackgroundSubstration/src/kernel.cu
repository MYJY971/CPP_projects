#include "hip/hip_runtime.h"
#include "kernel.cuh"

/*________________________________________________________* 
*														  *
*   		CUDA KERNELS AND ASSOCIATED FUNCTIONS		  *
*														  *
*_________________________________________________________*/

// device Kernel (can only be call by a kernel) :: define the modulo operation
inline __device__ int modulo(int val, int c){
	return (val & (c - 1));
}

// Kernel :: fill an image with a chekcerboard pattern
__global__ void _checkerboard(uchar4 *image, int step, uchar4 color1, uchar4 color2, unsigned int width, unsigned int height, unsigned int imStep)
{
	// get the position of the current pixel
	int x_local = blockIdx.x * blockDim.x + threadIdx.x;
	int y_local = blockIdx.y * blockDim.y + threadIdx.y;

	// exit if the pixel is out of the size of the image
	if (x_local >= width || y_local >= height) return;
	
	// fill the image, alternate the colors
	if (modulo(x_local, step) < (step/2))
		image[y_local * imStep + x_local] = modulo(y_local, step) < (step / 2) ? color1 : color2;
	else
		image[y_local * imStep + x_local] = modulo(y_local, step) < (step / 2) ? color2 : color1;
}

// Function :: fill an image with a chekcerboard pattern
void cuCreateCheckerboard(sl::zed::Mat &image)
{
	// get the image size
	unsigned int width = image.width;
	unsigned int height = image.height;

	// define the block dimension for the parallele computation
	dim3 dimGrid, dimBlock;
	dimBlock.x = 32;
	dimBlock.y = 8;

	dimGrid.x = ceill(width / (float)dimBlock.x);
	dimGrid.y = ceill(height / (float)dimBlock.y);

	// define the size of the square
	int step = 20;

	// define the two colors of the checkerboard
	uchar4 color1 = make_uchar4(250, 250, 250, 255);
	uchar4 color2 = make_uchar4(236, 172, 0, 255);
	
	// call the kernel
	_checkerboard << <dimGrid, dimBlock >> >((uchar4 *)image.data, step, color1, color2, width, height, image.step / sizeof(uchar4));
}

// Kernel :: replace the current image by an other if the depth if above the threshold
__global__ void _croppImage(float* depth, uchar4 * imageIn, uchar4 * imageOut, uchar4 * mask, float threshold,
	unsigned int width, unsigned int height, unsigned int depthStep, unsigned int imInStep, unsigned int imOutStep, unsigned int maskStep)
{
	// get the position of the current pixel
	int x_local = blockIdx.x * blockDim.x + threadIdx.x;
	int y_local = blockIdx.y * blockDim.y + threadIdx.y;

	// exit if the pixel is out of the size of the image
	if (x_local >= width || y_local >= height) return;

	// get the depth of the current pixel
	float D = depth[y_local * depthStep + x_local];

	// the depth is strickly positive, if not it means that the depth can not be computed
	// the depth should be below the threshold	
	if ((isfinite(D)) && (D < threshold))// keep the current image if true
		imageOut[y_local * imOutStep + x_local] = imageIn[y_local * imInStep + x_local];
	else // if false : replace current pixel by the pixel of the mask
		imageOut[y_local * imOutStep + x_local] = mask[y_local * maskStep + x_local];
}

// Function :: replace the current image by an other if the depth if above the threshold
void cuCroppImageByDepth(sl::zed::Mat &depth, sl::zed::Mat &imageLeft, sl::zed::Mat &imageCut, sl::zed::Mat &mask, float threshold)
{
	// get the image size
	unsigned int width = depth.width;
	unsigned int height = depth.height;

	// define the block dimension for the parallele computation
	dim3 dimGrid, dimBlock;
	dimBlock.x = 32;
	dimBlock.y = 8;

	dimGrid.x = ceill(width / (float)dimBlock.x);
	dimGrid.y = ceill(height / (float)dimBlock.y);
	
	// call the kernel
	_croppImage << <dimGrid, dimBlock >> >((float *)depth.data, (uchar4 *)imageLeft.data, (uchar4 *)imageCut.data, (uchar4 *)mask.data, threshold, width, height,
		depth.step / sizeof(float), imageLeft.step / sizeof(uchar4), imageCut.step / sizeof(uchar4), mask.step / sizeof(uchar4));
}